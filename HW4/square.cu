#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <sys/time.h>


#include <hip/hip_runtime.h>

#define N        10000000		// total number of items in vectors
#define nthreads 4	   // total number of threads in a block


__global__ void square(int n, int *vect1, int *vect2, int *sum)
{
	int threadID;
	threadID = blockIdx.x * blockDim.x + threadIdx.x;
	if(threadID < n)
	sum[threadID] = vect1[threadID] * vect1[threadID] + vect2[threadID] * vect2[threadID];
}

int main()
{			
	srand(time(NULL));	
	int *vect1_h, *vect2_h, *sum_h;
	int *vect1_d, *vect2_d, *sum_d;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	float milliseconds = 0;

	vect1_h = (int*)malloc( N* sizeof(int));
	vect2_h = (int*)malloc( N* sizeof(int));
	sum_h   = (int*)malloc( N* sizeof(int));
	
	hipMalloc((void**)&vect1_d, N * sizeof(int));
	hipMalloc((void**)&vect2_d, N * sizeof(int));
	hipMalloc((void**)&sum_d,   N * sizeof(int));

	for(int i = 0; i < N; i++)
	{	
		vect1_h[i] = rand()%10;
		vect2_h[i] = rand()%10;
	}

	hipMemcpy(vect1_d, vect1_h, N * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(vect2_d, vect2_h, N * sizeof(int), hipMemcpyHostToDevice);

	
	int nblocks = (N + nthreads - 1)/nthreads; 
		hipEventRecord(start);
	square<<<nblocks,nthreads>>>(N, vect1_d, vect2_d, sum_d);
    	hipEventRecord(stop);   	
	hipMemcpy(sum_h, sum_d, N * sizeof(int), hipMemcpyDeviceToHost);
	 
	 
	printf("Vector1: \n");	
	for(int i = 0; i < N; ++i)
		printf("  %d", vect1_h[i]);
	
	printf("\nVector2: \n");	
	for(int i = 0; i < N; ++i)
		printf("  %d", vect2_h[i]);

	
	printf("\nThe sum of squares of the vecors are is: \n");	
	for(int i = 0; i < N; ++i)
		printf("  %d", sum_h[i]);
	printf("\n");
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("elaspsed = %f ms", milliseconds);
	
	hipFree(vect1_d);
	hipFree(vect2_d);
	hipFree(sum_d);
	
	free(vect1_h);
	free(vect2_h);
	free(sum_h);
}
